// allocate and copy to memory of device
int arraySize = 1024;
int * d_in, * d_out;
hipMalloc(&d_in, sizeof(int)*arraySize);
hipMalloc(&d_out, sizeof(int));
hipMemcpy(d_in, h_in, sizeof(int)*arraySize, hipMemcpyHostToDevice);

// invoke kernel with the correct amount of threads and cache space
int numThreadsPerBlock = len;
int numBlocks = 1;
reduce <<< numBlocks, numThreadsPerBlock, sizeof(int)*numThreadsPerBlock >>> (d_in, d_out, len);

// copy result to host and free memory
hipMemcpy(h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
hipFree(d_in);
hipFree(d_out);
