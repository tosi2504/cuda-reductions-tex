// set the number of threads per block and calculate the required number of blocks
int arraySize = sizeof(int)*len;
int numThreadsPerBlock = 1024;
int numBlocks = (arraySize + numThreadsPerBlock - 1) / numThreadsPerBlock;

// allocate and copy into device memory
int * d_in, * d_out;
hipMalloc(&d_in, sizeof(int)*arraySize);
hipMalloc(&d_out, sizeof(int)*numBlocks);
hipMemcpy(d_in, h_in, sizeof(int)*arraySize, hipMemcpyHostToDevice);

// invoke kernel with the correct amount of threads and cache space
reduce <<< numBlocks, numThreadsPerBlock, sizeof(int)*numThreadsPerBlock >>> (d_in, d_out);

// copy result to host and free memory
hipMemcpy(h_out, d_out, sizeof(int)*numBlocks, hipMemcpyDeviceToHost);
hipFree(d_in);
hipFree(d_out);
